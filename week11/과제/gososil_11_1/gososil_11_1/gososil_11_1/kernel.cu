#include "hip/hip_runtime.h"
﻿#include "hip/hip_runtime.h"
#include ""

#include <stdio.h>
#include<stdio.h>
#include<stdlib.h>
#include <math.h>
#include <Windows.h>
#include <time.h>
#include <assert.h>

#define CUDA_CALL(x) { const hipError_t a = (x); if(a != hipSuccess) { printf("\nCuda Error: %s (err_num=%d) at line:%d\n", hipGetErrorString(a), a, __LINE__); hipDeviceReset(); assert(0);}}
typedef float TIMER_T;
#define USE_CPU_TIMER 1
#define USE_GPU_TIMER 1

#define	IN
#define OUT
#define INOUT

#if USE_CPU_TIMER == 1
__int64 start, freq, end;
#define CHECK_TIME_START { QueryPerformanceFrequency((LARGE_INTEGER*)&freq); QueryPerformanceCounter((LARGE_INTEGER*)&start); }
#define CHECK_TIME_END(a) { QueryPerformanceCounter((LARGE_INTEGER*)&end); a = (float)((float)(end - start) / (freq / 1000.0f)); }
#else
#define CHECK_TIME_START
#define CHECK_TIME_END(a)
#endif

#if USE_GPU_TIMER == 1
hipEvent_t cuda_timer_start, cuda_timer_stop;
#define CUDA_STREAM_0 (0)

void create_device_timer()
{
	CUDA_CALL(hipEventCreate(&cuda_timer_start));
	CUDA_CALL(hipEventCreate(&cuda_timer_stop));
}

void destroy_device_timer()
{
	CUDA_CALL(hipEventDestroy(cuda_timer_start));
	CUDA_CALL(hipEventDestroy(cuda_timer_stop));
}

inline void start_device_timer()
{
	hipEventRecord(cuda_timer_start, CUDA_STREAM_0);
}

inline TIMER_T stop_device_timer()
{
	TIMER_T ms;
	hipEventRecord(cuda_timer_stop, CUDA_STREAM_0);
	hipEventSynchronize(cuda_timer_stop);

	hipEventElapsedTime(&ms, cuda_timer_start, cuda_timer_stop);
	return ms;
}

#define CHECK_TIME_INIT_GPU() { create_device_timer(); }
#define CHECK_TIME_START_GPU() { start_device_timer(); }
#define CHECK_TIME_END_GPU(a) { a = stop_device_timer(); }
#define CHECK_TIME_DEST_GPU() { destroy_device_timer(); }
#else
#define CHECK_TIME_INIT_GPU()
#define CHECK_TIME_START_GPU()
#define CHECK_TIME_END_GPU(a)
#define CHECK_TIME_DEST_GPU()
#endif

#define N_EQUATIONS (1 << 26)	// 전체 데이터 사이즈

#define BLOCK_SIZE (1 << 6)	// CUDA 커널 thread block 사이즈
#define BLOCK_WIDTH (1 << 3)
#define BLOCK_HEIGHT (BLOCK_SIZE / BLOCK_WIDTH)

#define N_ITERATION (1 << 0)	// 실험 반복 횟수

#define FLOAT_ERR 0.01	// 실험 결과 0 판별용

TIMER_T compute_time = 0;
TIMER_T device_time = 0;

int N;

float* A;
float* B;
float* C;
float* X0_CPU;
float* X0_GPU;
float* X1_CPU;
float* X1_GPU;
float* FX0_CPU;
float* FX0_GPU;
float* FX1_CPU;
float* FX1_GPU;

hipError_t find_roots_GPU(float* A, float* B, float* C,
	float* X0, float* X1, float* FX0, float* FX1, int n);

__global__ void find_roots_Kernel(float* A, float* B, float* C,
	float* X0, float* X1, float* FX0, float* FX1) {
	int row, col, i;
	float a, b, c, d, x0, x1, tmp;
	row = blockDim.y * blockIdx.y + threadIdx.y;
	col = blockDim.x * blockIdx.x + threadIdx.x;
	i = gridDim.x * blockDim.x * row + col;
	a = A[i]; b = B[i]; c = C[i];
	d = sqrtf(b * b - 4.0f * a * c);
	tmp = 1.0f / (2.0f * a);
	X0[i] = x0 = (-b - d) * tmp;
	X1[i] = x1 = (-b + d) * tmp;
	FX0[i] = (a * x0 + b) * x0 + c;
	FX1[i] = (a * x1 + b) * x1 + c;
}

void find_roots_CPU(float* A, float* B, float* C,
	float* X0, float* X1, float* FX0, float* FX1, int n) {
	int i;
	float a, b, c, d, x0, x1, tmp;
	for (i = 0; i < n; i++) {
		a = A[i]; b = B[i]; c = C[i];
		d = sqrtf(b * b - 4.0f * a * c);
		tmp = 1.0f / (2.0f * a);
		X0[i] = x0 = (-b - d) * tmp;
		X1[i] = x1 = (-b + d) * tmp;
		FX0[i] = (a * x0 + b) * x0 + c;
		FX1[i] = (a * x1 + b) * x1 + c;
	}
}

void read_bin_file() {
	printf("**Binary File Read Create..\n");
	//A.bin B.bin 파일을 읽는다
	// 첫 4bytes는 전체 데이터 개수, 다음 4bytes는 Nf의 크기, 그 이후 N개의 int형 데이터저장
	//데이터는 [-100, 100] 구간의 정수
	FILE* fp = fopen("A.bin", "rb");
	A = (float*)malloc(N * sizeof(float));
	fread(A, sizeof(float), N, fp);
	fclose(fp);

	fp = fopen("B.bin", "rb");
	B = (float*)malloc(N * sizeof(float));
	fread(B, sizeof(float), N, fp);
	fclose(fp);

	fp = fopen("C.bin", "rb");
	C = (float*)malloc(N * sizeof(float));
	fread(C, sizeof(float), N, fp);
	fclose(fp);
	printf("**Binary File Read Complete!\n\n");
}

void init_bin_file(IN int n) {
	printf("**Binary File Create Create..\n");
	srand((unsigned)time(NULL));
	FILE* fp1 = fopen("A.bin", "wb");
	FILE* fp2 = fopen("B.bin", "wb");
	FILE* fp3 = fopen("C.bin", "wb");

	int i;
	float a, b, c;
	for (i = 0; i < n; i++) {
		while (1) {
			a = (float)((float)rand() / RAND_MAX * 200.0 - 100.0);
			b = (float)((float)rand() / RAND_MAX * 200.0 - 100.0);
			c = (float)((float)rand() / RAND_MAX * 200.0 - 100.0);
			if (b * b > 4.0 * a * c + 0.01 && b * b - 4.0 * a * c > 10.0)
				break;
		}
		fwrite(&a, sizeof(float), 1, fp1);
		fwrite(&b, sizeof(float), 1, fp2);
		fwrite(&c, sizeof(float), 1, fp3);
	}
	fclose(fp1);
	fclose(fp2);
	fclose(fp3);
	printf("**Binary File Create Complete!\n\n");
}
void save_ans(float* X0, float* X1, float* FX0, float* FX1, int n) {
	printf("**Answer Save Create..\n");
	FILE* fp = fopen("X0.bin", "wb");
	for (int i = 0; i < n; i++) {
		fwrite(&X0[i], sizeof(float), 1, fp);
	}
	fclose(fp);

	fp = fopen("X1.bin", "wb");
	for (int i = 0; i < n; i++) {
		fwrite(&X1[i], sizeof(float), 1, fp);
	}
	fclose(fp);

	fp = fopen("FX0.bin", "wb");
	for (int i = 0; i < n; i++) {
		fwrite(&FX0[i], sizeof(float), 1, fp);
	}
	fclose(fp);

	fp = fopen("FX1.bin", "wb");
	for (int i = 0; i < n; i++) {
		fwrite(&FX1[i], sizeof(float), 1, fp);
	}
	fclose(fp);
	printf("**Answer Save Complete!\n\n");
}
int main()
{
	int i;
	N = N_EQUATIONS;
	init_bin_file(N);
	read_bin_file();

	TIMER_T CPU_time = 0.0f, GPU_time_NO_SHARED = 0.0f;
	X0_CPU = (float*)malloc(N * sizeof(float));
	X0_GPU = (float*)malloc(N * sizeof(float));
	X1_CPU = (float*)malloc(N * sizeof(float));
	X1_GPU = (float*)malloc(N * sizeof(float));
	FX0_CPU = (float*)malloc(N * sizeof(float));
	FX0_GPU = (float*)malloc(N * sizeof(float));
	FX1_CPU = (float*)malloc(N * sizeof(float));
	FX1_GPU = (float*)malloc(N * sizeof(float));
	for (i = 0; i < N_ITERATION; i++) {
		CHECK_TIME_START;
		find_roots_CPU(A, B, C, X0_CPU, X1_CPU, FX0_CPU, FX1_CPU, N);
		CHECK_TIME_END(compute_time);
		CPU_time += compute_time;
		find_roots_GPU(A, B, C, X0_GPU, X1_GPU, FX0_GPU, FX1_GPU, N);
		GPU_time_NO_SHARED += device_time;
	}
	
	for (i = 0; i < N; i++) {
		if (X0_CPU[i] - X0_GPU[i] > FLOAT_ERR) {
			printf("%d : CPU : %d,\tGPU : %d\n", i, X0_CPU[i], X0_GPU[i]);
			break;
		}
	}
	if (i == N)
		printf("**CPU answer and Kernel answer same\n\n");
	else
		printf("**CPU answer and Kernel answer different!!\n\n");

	printf("**N_EQUATIONS : %d BLOCK_SIZE : %d\n", N_EQUATIONS, BLOCK_SIZE);
	printf("**CPU compute time : %f ms\n", CPU_time / N_ITERATION);
	printf("**GPU NO SHARED compute time : %f ms\n", GPU_time_NO_SHARED / N_ITERATION);
	save_ans(X0_GPU, X1_GPU, FX0_GPU, FX1_GPU, N);
	free(A);
	free(B);
	free(C);
	free(X0_CPU);
	free(X0_GPU);
	free(X1_CPU);
	free(X1_GPU);
	free(FX0_CPU);
	free(FX0_GPU);
	free(FX1_CPU);
	free(FX1_GPU);

	return 0;
}

////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////
//	
//	커널을 실행하기 전 필요한 자료들 준비 및 커널을 실행할 디바이스를 설정
//	
////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////

hipError_t find_roots_GPU(float* A, float* B, float* C,
	float* X0, float* X1, float* FX0, float* FX1, int n) {
	//커널 실행전 필요 자료 준비 밎 커널 실행
	hipError_t cudaStatus;
	CHECK_TIME_INIT_GPU();
	cudaStatus = hipSetDevice(0);
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipSetDevice failed!  Do you have a CUDA-capable GPU installed?");
		return cudaStatus;
	}
	float* _A;
	float* _B;
	float* _C;
	float* _X0;
	float* _X1;
	float* _FX0;
	float* _FX1;
	size_t size;
	size = N * sizeof(float);
	CUDA_CALL(hipMalloc(&_A, size))
	CUDA_CALL(hipMemcpy(_A, A, size, hipMemcpyHostToDevice))
	CUDA_CALL(hipMalloc(&_B, size))
	CUDA_CALL(hipMemcpy(_B, B, size, hipMemcpyHostToDevice))
	CUDA_CALL(hipMalloc(&_C, size))
	CUDA_CALL(hipMemcpy(_C, C, size, hipMemcpyHostToDevice))
	CUDA_CALL(hipMalloc(&_X0, size))
	CUDA_CALL(hipMalloc(&_X1, size))
	CUDA_CALL(hipMalloc(&_FX0, size))
	CUDA_CALL(hipMalloc(&_FX1, size))
	dim3 dimBlock(BLOCK_WIDTH, BLOCK_HEIGHT);
	dim3 dimGrid((1 << 13) / BLOCK_WIDTH, (1 << 13) / BLOCK_HEIGHT);
	CHECK_TIME_START_GPU()
	find_roots_Kernel << < dimGrid, dimBlock >> > (_A, _B, _C, _X0, _X1, _FX0, _FX1);
	CHECK_TIME_END_GPU(device_time)
	CUDA_CALL(hipGetLastError())
	CUDA_CALL(hipDeviceSynchronize())
	CUDA_CALL(hipMemcpy(X0, _X0, size, hipMemcpyDeviceToHost))
	CUDA_CALL(hipMemcpy(X1, _X1, size, hipMemcpyDeviceToHost))
	CUDA_CALL(hipMemcpy(FX0, _FX0, size, hipMemcpyDeviceToHost))
	CUDA_CALL(hipMemcpy(FX1, _FX1, size, hipMemcpyDeviceToHost))
	hipFree(_A);
	hipFree(_B);
	hipFree(_C);
	hipFree(_X0);
	hipFree(_X1);
	hipFree(_FX0);
	hipFree(_FX1);
	////
	
	return cudaStatus;
}
