#include "hip/hip_runtime.h"
#pragma once

#include "cuda_code.cuh"

#include <stdio.h>
#include<stdio.h>
#include<stdlib.h>
#include <math.h>
#include <Windows.h>
#include <time.h>
#include <assert.h>

#if USE_CPU_TIMER == 1
__int64 start, freq, end;
#define CHECK_TIME_START { QueryPerformanceFrequency((LARGE_INTEGER*)&freq); QueryPerformanceCounter((LARGE_INTEGER*)&start); }
#define CHECK_TIME_END(a) { QueryPerformanceCounter((LARGE_INTEGER*)&end); a = (float)((float)(end - start) / (freq / 1000.0f)); }
#else
#define CHECK_TIME_START
#define CHECK_TIME_END(a)
#endif

#if USE_GPU_TIMER == 1
hipEvent_t cuda_timer_start, cuda_timer_stop;
#define CUDA_STREAM_0 (0)

void create_device_timer()
{
	CUDA_CALL(hipEventCreate(&cuda_timer_start));
	CUDA_CALL(hipEventCreate(&cuda_timer_stop));
}

void destroy_device_timer()
{
	CUDA_CALL(hipEventDestroy(cuda_timer_start));
	CUDA_CALL(hipEventDestroy(cuda_timer_stop));
}

inline void start_device_timer()
{
	hipEventRecord(cuda_timer_start, CUDA_STREAM_0);
}

inline TIMER_T stop_device_timer()
{
	TIMER_T ms;
	hipEventRecord(cuda_timer_stop, CUDA_STREAM_0);
	hipEventSynchronize(cuda_timer_stop);

	hipEventElapsedTime(&ms, cuda_timer_start, cuda_timer_stop);
	return ms;
}

#define CHECK_TIME_INIT_GPU() { create_device_timer(); }
#define CHECK_TIME_START_GPU() { start_device_timer(); }
#define CHECK_TIME_END_GPU(a) { a = stop_device_timer(); }
#define CHECK_TIME_DEST_GPU() { destroy_device_timer(); }
#else
#define CHECK_TIME_INIT_GPU()
#define CHECK_TIME_START_GPU()
#define CHECK_TIME_END_GPU(a)
#define CHECK_TIME_DEST_GPU()
#endif

TIMER_T compute_time = 0;
TIMER_T device_time = 0;

#define Window 2

__constant__ float constant_gaussian_kernel[ 25 ];

////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////
//	
//	Gaussian ���͸��� �ϴ� Ŀ��
//	shared memory�� ������� �ʴ´�
//	
////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////

__global__ void Gaussian_kernel_no_shared(IN unsigned char *d_bitmaps, OUT unsigned char *d_Gaussian, long width, long height) {	
	int x = blockDim.x * blockIdx.x + threadIdx.x;
	int y = blockDim.y * blockIdx.y + threadIdx.y;
	int tid = gridDim.x * blockDim.x * y + x;
	int i,j,k;
	double mean =0.0;

	for (int l = -2; l <= 2; l++) {
		for (int k = -2; k <= 2; k++) {
			if ((y + l) < 0 || (y + l) >= height || (x + k) < 0 || (x + k) >= width) 
				continue;
			mean += d_bitmaps[(y + l) * gridDim.x * blockDim.x + (x + k)] * constant_gaussian_kernel[(l + 2) * 5 + (k + 2)];
		}
	}
	d_Gaussian[tid] = mean;
}

////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////
//	
//	Gaussian ���͸��� �ϴ� Ŀ��
//	shared memory�� ����Ѵ�.
//	
////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////
extern __shared__ unsigned char sharedBuffer[];
__global__ void Gaussian_kernel_shared(INOUT unsigned char *d_bitmaps, OUT unsigned char *d_Gaussian, long width, long height) {
	int row = blockDim.y * blockIdx.y + threadIdx.y;
	int col = blockDim.x * blockIdx.x + threadIdx.x;
	int x = threadIdx.x + 2;
	int y = threadIdx.y + 2;
	int id, tid;
	if (x == 2) {
		if (col > 0) {
			id = (blockDim.x + 4) * y + (x - 1);
			tid = gridDim.x * blockDim.x * row + (col - 1);
			sharedBuffer[id] = d_bitmaps[tid];
		}
		if (col > 1) {
			id = (blockDim.x + 4) * y + (x - 2);
			tid = gridDim.x * blockDim.x * row + (col - 2);
			sharedBuffer[id] = d_bitmaps[tid];
		}
	}
	if (y == 2) {
		if (row > 0) {
			id = (blockDim.x + 4) * (y - 1) + (x);
			tid = gridDim.x * blockDim.x * (row - 1) + col;
			sharedBuffer[id] = d_bitmaps[tid];
		}
		if (row > 1) {
			id = (blockDim.x + 4) * (y - 2) + (x);
			tid = gridDim.x * blockDim.x * (row - 2) + col;
			sharedBuffer[id] = d_bitmaps[tid];
		}
	}
	if (x == 2 && y == 2) {
		for (int i = 1; i <= 2; i++) {
			for (int j = 1; j <= 2; j++) {
				if (row - i < 0) continue;
				if (col - j < 0) continue;
				id = (blockDim.x + 4) * (y - i) + (x - j);
				tid = gridDim.x * blockDim.x * (row - i) + (col - j);
				sharedBuffer[id] = d_bitmaps[tid];
			}

		}
	}
	if (x == blockDim.x + 1) {
		if (col + 1 < width) {
			id = (blockDim.x + 4) * y + (x + 1);
			tid = gridDim.x * blockDim.x * row + (col + 1);
			sharedBuffer[id] = d_bitmaps[tid];
		}
		if (col + 2 < width) {
			id = (blockDim.x + 4) * y + (x + 2);
			tid = gridDim.x * blockDim.x * row + (col + 2);
			sharedBuffer[id] = d_bitmaps[tid];
		}
	}
	if (y == blockDim.y + 1) {
		if (row + 1 < height) {
			id = (blockDim.x + 4) * (y + 1) + (x);
			tid = gridDim.x * blockDim.x * (row + 1) + (col);
			sharedBuffer[id] = d_bitmaps[tid];
		}
		if (row + 2 < height) {
			id = (blockDim.x + 4) * (y + 2) + (x);
			tid = gridDim.x * blockDim.x * (row + 2) + (col);
			sharedBuffer[id] = d_bitmaps[tid];
		}
	}
	if (x == blockDim.x + 1 && y == blockDim.y + 1) {
		for (int i = 1; i <= 2; i++) {
			for (int j = 1; j <= 2; j++) {
				if (row + i >= height) continue;
				if (col + j >= width) continue;
				id = (blockDim.x + 4) * (y + i) + (x + j);
				tid = gridDim.x * blockDim.x * (row + i) + (col + j);
				sharedBuffer[id] = d_bitmaps[tid];
			}

		}
	}
	id = (blockDim.x + 4) * y + x;
	tid = gridDim.x * blockDim.x * row + col;
	sharedBuffer[id] = d_bitmaps[tid];
	__syncthreads();
	double mean = 0;
	for (int l = -2; l <= 2; l++) {
		for (int k = -2; k <= 2; k++) {
			if ((row + l) < 0 || (row + l) >= height || (col + k) < 0 || (col + k) >= width) continue;
			mean += sharedBuffer[(y + l) * (blockDim.x + 4) + (x + k)] * constant_gaussian_kernel[(l + 2) * 5 + (k + 2)];
		}
	}
	d_Gaussian[tid] = mean;
}

////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////
//	
//	Constant variable �� gaussian kernel�� �����ϴ� �Լ�
//	�Ŀ� gaussian filtering ���� ����Ѵ�.
//	
////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////

void Set_Gaussain_Kernel(){
	float _1 = 1.0f / 256.0f;
	float _4 = _1 * 4;
	float _6 = _1 * 6;
	float _16 = _1 * 16;
	float _24 = _1 * 24;
	float _36 = _1 * 36;

	float *p_gaussian_kernel = new float[25];

	p_gaussian_kernel[0] = p_gaussian_kernel[4] = p_gaussian_kernel[20] = p_gaussian_kernel[24] = _1;
	p_gaussian_kernel[1] = p_gaussian_kernel[3] = p_gaussian_kernel[5] = p_gaussian_kernel[9]= _4;
	p_gaussian_kernel[15] = p_gaussian_kernel[19] = p_gaussian_kernel[21] = p_gaussian_kernel[23] = _4;
	p_gaussian_kernel[2] = p_gaussian_kernel[10] = p_gaussian_kernel[14] = p_gaussian_kernel[22] = _6;
	p_gaussian_kernel[6] = p_gaussian_kernel[8] = p_gaussian_kernel[16] = p_gaussian_kernel[18] = _16;
	p_gaussian_kernel[7] = p_gaussian_kernel[11] =p_gaussian_kernel[13] = p_gaussian_kernel[17] = _24;
	p_gaussian_kernel[12] = _36;

	hipMemcpyToSymbol(HIP_SYMBOL( constant_gaussian_kernel), p_gaussian_kernel, sizeof( float ) * 25 );

	delete[] p_gaussian_kernel;
}

////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////
//	
//	Ŀ���� �����ϱ� �� �ʿ��� �ڷ�� �غ� �� Ŀ���� ������ ����̽��� ����
//	Shared_flag �Է� �� NO_SHARED �� SHARED �� �� ���� ��ũ�θ� ������
//	flag���� �´� Ŀ���� ����
//	
////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////

float Do_Gaussian_on_GPU(IN unsigned char *p_bitmaps, OUT unsigned char *p_Gaussian, long width, long height, int Shared_flag)
{
	Set_Gaussain_Kernel();
	CUDA_CALL(hipSetDevice(0));
	unsigned int total_pixel = width * height;

	unsigned char *d_bitmaps, *d_Gaussian;
	size_t mem_size;

	mem_size = width * height * sizeof(unsigned char);
	CUDA_CALL(hipMalloc(&d_bitmaps, mem_size));

	CUDA_CALL(hipMemcpy(d_bitmaps, p_bitmaps, mem_size, hipMemcpyHostToDevice));

	CUDA_CALL(hipMalloc(&d_Gaussian, mem_size));

	dim3 blockDim(1024, 1024);

	//dim3 gridDim((width + 31) / blockDim.x, (height + 31) / blockDim.y);

	dim3 gridDim((width + 1023) / blockDim.x, (height + 1023) / blockDim.y);
	CHECK_TIME_INIT_GPU();
	CHECK_TIME_START_GPU();

	switch (Shared_flag)
	{
	case NO_SHARED:
		Gaussian_kernel_no_shared << <gridDim , blockDim>> > (d_bitmaps, d_Gaussian, width, height);
		break;
	case SHARED:
		Gaussian_kernel_shared << < gridDim, blockDim, sizeof(unsigned char) * (blockDim.x + 2 * Window) * (blockDim.y + 2 * Window)>> > (d_bitmaps, d_Gaussian, width, height);
		break;
	}

	CUDA_CALL(hipDeviceSynchronize());
	CHECK_TIME_END_GPU(device_time);
	CHECK_TIME_DEST_GPU();

	CUDA_CALL(hipMemcpy(p_Gaussian, d_Gaussian, mem_size, hipMemcpyDeviceToHost));
	
	hipFree(d_bitmaps);
	hipFree(d_Gaussian);

	return device_time;
}